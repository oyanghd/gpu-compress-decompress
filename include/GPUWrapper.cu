#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "parallelHeader.h"
__constant__ unsigned char d_bitSequenceConstMemory[256][255];

void lauchCUDAHuffmanCompress(unsigned char *inputFileData, unsigned int *compressedDataOffset, unsigned int inputFileLength, int numKernelRuns, unsigned int integerOverflowFlag, long unsigned int mem_req){
	int i;
	unsigned char *d_inputFileData, *d_byteCompressedData;
	unsigned int *d_compressedDataOffset;
	struct huffmanDictionary *d_huffmanDictionary;
	unsigned int *gpuBitPaddingFlag, *bitPaddingFlag;
	unsigned int *gpuMemoryOverflowIndex, *integerOverflowIndex;
	long unsigned int mem_free, mem_total;
	hipError_t error;
	
	// generate offset 
	if(integerOverflowFlag == 0){
		// only one time run of kernel
		if(numKernelRuns == 1){
			createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength);
		}
		
		// multiple run of kernel due to larger file or smaller gpu memory
		else{
			gpuBitPaddingFlag = (unsigned int *)calloc(numKernelRuns, sizeof(unsigned int));
			gpuMemoryOverflowIndex = (unsigned int *)calloc(numKernelRuns * 2, sizeof(unsigned int));
			createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength, gpuMemoryOverflowIndex, gpuBitPaddingFlag, mem_req);
		}
	}
	
	// when there is integer over flow
	else{
		// overflow occurs and single run
		if(numKernelRuns == 1){
			bitPaddingFlag = (unsigned int *)calloc(numKernelRuns, sizeof(unsigned int));
			integerOverflowIndex = (unsigned int *)calloc(numKernelRuns * 2, sizeof(unsigned int));
			createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength, integerOverflowIndex, bitPaddingFlag, 10240);
		}
		
		// overflow occurs and multiple run
		else{
			gpuBitPaddingFlag = (unsigned int *)calloc(numKernelRuns, sizeof(unsigned int));
			bitPaddingFlag = (unsigned int *)calloc(numKernelRuns, sizeof(unsigned int));
			integerOverflowIndex = (unsigned int *)calloc(numKernelRuns * 2, sizeof(unsigned int));
			gpuMemoryOverflowIndex = (unsigned int *)calloc(numKernelRuns * 2, sizeof(unsigned int));
			createDataOffsetArray(compressedDataOffset, inputFileData, inputFileLength, integerOverflowIndex, bitPaddingFlag, gpuMemoryOverflowIndex, gpuBitPaddingFlag, 10240, mem_req);	
		}
	}
	
	// GPU initiation
	{	
		// allocate memory for input data, offset information and dictionary
		error = hipMalloc((void **)&d_inputFileData, inputFileLength * sizeof(unsigned char));
		if (error != hipSuccess)
				printf("erro_1: %s\n", hipGetErrorString(error));
			
		error = hipMalloc((void **)&d_compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int));
		if (error != hipSuccess)
				printf("erro_2: %s\n", hipGetErrorString(error));
		error = hipMalloc((void **)&d_huffmanDictionary, sizeof(huffmanDictionary));
		if (error != hipSuccess)
				printf("erro_3: %s\n", hipGetErrorString(error));
			
		// memory copy input data, offset information and dictionary
		error = hipMemcpy(d_inputFileData, inputFileData, inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (error!= hipSuccess)
				printf("erro_4: %s\n", hipGetErrorString(error));
		error = hipMemcpy(d_compressedDataOffset, compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
		if (error!= hipSuccess)
				printf("erro_5: %s\n", hipGetErrorString(error));
		error = hipMemcpy(d_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDictionary), hipMemcpyHostToDevice);
		if (error!= hipSuccess)
				printf("erro_6: %s\n", hipGetErrorString(error));
			
		// copy constant memory if required for dictionary
		if(constMemoryFlag == 1){
			error = hipMemcpyToSymbol(HIP_SYMBOL(d_bitSequenceConstMemory), bitSequenceConstMemory, 256 * 255 * sizeof(unsigned char));
			if (error!= hipSuccess)
				printf("erro_const: %s\n", hipGetErrorString(error));
		}
	}

	
	// memory copy of offset data
	if(numKernelRuns == 1){
		// no overflow
		if(integerOverflowFlag == 0){
			error = hipMalloc((void **)&d_byteCompressedData, (compressedDataOffset[inputFileLength]) * sizeof(unsigned char));
			if (error!= hipSuccess)
				printf("erro_7: %s\n", hipGetErrorString(error));
			
			// initialize d_byteCompressedData 
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
			
			// debug
			// if(1){
			// 	hipMemGetInfo(&mem_free, &mem_total);
			// 	printf("Free Mem: %lu\n", mem_free);		
			// }			
			
			// run kernel
			compress<<<1, BLOCK_SIZE>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, inputFileLength, constMemoryFlag);
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));
			
			// copy compressed data from GPU to CPU memory
			error = hipMemcpy(inputFileData, d_inputFileData, ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
			if (error != hipSuccess)
				printf("erro_9: %s\n", hipGetErrorString(error));
			
			// free allocated memory
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
		}
		
		// integer overflow occurs
		else{
			// additional variable to store offset data after integer oveflow
			unsigned char *d_byteCompressedDataOverflow;
			
			// allocate memory to store offset information
			error = hipMalloc((void **)&d_byteCompressedData, compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_7: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_byteCompressedDataOverflow, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
			
			// initialize offset data
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_9: %s\n", hipGetErrorString(error));	
			error = hipMemset(d_byteCompressedDataOverflow, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_10: %s\n", hipGetErrorString(error));
			
			// debug
			// if(1){
			// 	hipMemGetInfo(&mem_free, &mem_total);
			// 	printf("Free Mem: %lu\n", mem_free);		
			// }
			
			// launch kernel
			compress<<<1, BLOCK_SIZE>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, d_byteCompressedDataOverflow, inputFileLength, constMemoryFlag, integerOverflowIndex[0]);
			
			// check status
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));
			
			// get output data
			if(bitPaddingFlag[0] == 0){
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex[0]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));
			}
			else{
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex[0]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				unsigned char temp_compByte = inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1];
				
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));				
				inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1] = temp_compByte | inputFileData[(compressedDataOffset[integerOverflowIndex[0]] / 8) - 1];
			}

			// free allocated memory
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
			hipFree(d_byteCompressedDataOverflow);
		}
	}
	
	else{
		if(integerOverflowFlag == 0){
			error = hipMalloc((void **)&d_byteCompressedData, (compressedDataOffset[gpuMemoryOverflowIndex[1]]) * sizeof(unsigned char));
			if (error!= hipSuccess)
				printf("erro_7: %s\n", hipGetErrorString(error));

			// debug
			// if(1){
			// 	hipMemGetInfo(&mem_free, &mem_total);
			// 	printf("Free Mem: %lu\n", mem_free);		
			// }		
			
			unsigned int pos = 0;
			for(i = 0; i < numKernelRuns; i++){
				// initialize d_byteCompressedData 
				error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
				if (error!= hipSuccess)
						printf("erro_8: %s\n", hipGetErrorString(error));
	
				compress<<<1, BLOCK_SIZE>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, gpuMemoryOverflowIndex[i * 2], constMemoryFlag, gpuMemoryOverflowIndex[i * 2 + 1]);
				hipError_t error_kernel = hipGetLastError();
				if (error_kernel != hipSuccess)
					printf("erro_final: %s\n", hipGetErrorString(error_kernel));	
	
	
				if(gpuBitPaddingFlag[i] == 0){
					error = hipMemcpy(&inputFileData[pos], d_inputFileData, (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
					if (error != hipSuccess)
						printf("erro_12: %s\n", hipGetErrorString(error));
					pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
				}
				else{
					unsigned char temp_compByte = inputFileData[pos - 1];
					error = hipMemcpy(&inputFileData[pos - 1], d_inputFileData, ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
					if (error != hipSuccess)
						printf("erro_12: %s\n", hipGetErrorString(error));
					inputFileData[pos - 1] = temp_compByte | inputFileData[pos - 1];
					pos +=  (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
				}
			}
	
			
			// free allocated memory
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
		}
		
		else{
			// additional variable to store offset data after integer oveflow
			unsigned char *d_byteCompressedDataOverflow;
			error = hipMalloc((void **)&d_byteCompressedData, (compressedDataOffset[integerOverflowIndex[0]]) * sizeof(unsigned char));
			if (error!= hipSuccess)
				printf("erro_7: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_byteCompressedDataOverflow, compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));

			// debug
			// if(1){
			// 	hipMemGetInfo(&mem_free, &mem_total);
			// 	printf("Free Mem: %lu\n", mem_free);		
			// }		
			
			unsigned int pos = 0;
			for(i = 0; i < numKernelRuns; i++){
				if(integerOverflowIndex[i] != 0){
					// initialize d_byteCompressedData 
					error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
					if (error!= hipSuccess)
							printf("erro_9: %s\n", hipGetErrorString(error));
					error = hipMemset(d_byteCompressedDataOverflow, 0, compressedDataOffset[gpuMemoryOverflowIndex[1]] * sizeof(unsigned char));
					if (error!= hipSuccess)
							printf("erro_10: %s\n", hipGetErrorString(error));
		
					compress<<<1, BLOCK_SIZE>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, d_byteCompressedDataOverflow, gpuMemoryOverflowIndex[i * 2], constMemoryFlag, gpuMemoryOverflowIndex[i * 2 + 1], integerOverflowIndex[i]);
					hipError_t error_kernel = hipGetLastError();
					if (error_kernel != hipSuccess)
						printf("erro_final: %s\n", hipGetErrorString(error_kernel));	
		
					if(gpuBitPaddingFlag[i] == 0){
						if(bitPaddingFlag[i] == 0){
							error = hipMemcpy(&inputFileData[pos], d_inputFileData, (compressedDataOffset[integerOverflowIndex[i]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_11: %s\n", hipGetErrorString(error));
							error = hipMemcpy(&inputFileData[pos + (compressedDataOffset[integerOverflowIndex[i]] / 8)], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8)], ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_12: %s\n", hipGetErrorString(error));
							pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) + (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
						}
						else{
							error = hipMemcpy(&inputFileData[pos], d_inputFileData, (compressedDataOffset[integerOverflowIndex[i]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_11: %s\n", hipGetErrorString(error));
							unsigned char temp_compByte = inputFileData[pos + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1];
							
							error = hipMemcpy(&inputFileData[pos + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8)], ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_12: %s\n", hipGetErrorString(error));				
							inputFileData[pos + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1] = temp_compByte | inputFileData[pos + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1];
							pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) + (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
						}
					}
					else{
						unsigned char temp_gpuCompByte = inputFileData[pos - 1];
						if(bitPaddingFlag[i] == 0){
							error = hipMemcpy(&inputFileData[pos - 1], d_inputFileData, (compressedDataOffset[integerOverflowIndex[i]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_11: %s\n", hipGetErrorString(error));
							error = hipMemcpy(&inputFileData[pos -1 + (compressedDataOffset[integerOverflowIndex[i]] / 8)], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8)], ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_12: %s\n", hipGetErrorString(error));
							inputFileData[pos - 1] = temp_gpuCompByte | inputFileData[pos - 1];
							pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) + (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
						}
						else{
							error = hipMemcpy(&inputFileData[pos - 1], d_inputFileData, (compressedDataOffset[integerOverflowIndex[i]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_11: %s\n", hipGetErrorString(error));
							unsigned char temp_compByte = inputFileData[ pos -1 + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1];
							
							error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8) - 1], &d_inputFileData[(compressedDataOffset[integerOverflowIndex[i]] / 8)], ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
							if (error != hipSuccess)
								printf("erro_12: %s\n", hipGetErrorString(error));				
							inputFileData[(compressedDataOffset[pos - 1 + integerOverflowIndex[i]] / 8) - 1] = temp_compByte | inputFileData[pos - 1 + (compressedDataOffset[integerOverflowIndex[i]] / 8) - 1];
							inputFileData[pos - 1] = temp_gpuCompByte | inputFileData[pos - 1];
							pos += (compressedDataOffset[integerOverflowIndex[i]] / 8) + (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 2;
						}
					}			
				}
				else{
					// initialize d_byteCompressedData
					error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[integerOverflowIndex[0]] * sizeof(unsigned char));
					if (error!= hipSuccess)
							printf("erro_8: %s\n", hipGetErrorString(error));
		
					compress<<<1, BLOCK_SIZE>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, gpuMemoryOverflowIndex[i * 2], constMemoryFlag, gpuMemoryOverflowIndex[i * 2 + 1]);
					hipError_t error_kernel = hipGetLastError();
					if (error_kernel != hipSuccess)
						printf("erro_final: %s\n", hipGetErrorString(error_kernel));	
		
		
					if(gpuBitPaddingFlag[i] == 0){
						error = hipMemcpy(&inputFileData[pos], d_inputFileData, (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
						if (error != hipSuccess)
							printf("erro_12: %s\n", hipGetErrorString(error));
						pos += (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8);
					}
					else{
						unsigned char temp_huffmanTreeNode = inputFileData[pos - 1];
						error = hipMemcpy(&inputFileData[pos - 1], d_inputFileData, ((compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
						if (error != hipSuccess)
							printf("erro_12: %s\n", hipGetErrorString(error));
						inputFileData[pos - 1] = temp_huffmanTreeNode | inputFileData[pos - 1];
						pos +=  (compressedDataOffset[gpuMemoryOverflowIndex[i * 2 + 1]] / 8) - 1;
					}			
				}
			}
	
			// free allocated memory
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);		
		}
	}
}